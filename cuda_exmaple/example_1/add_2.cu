#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>

#define CUDA_ERROR_CHECK
#include "/home/wxie/AI/Spike/July_2019/Spike_July_2019/Spike/Backend/CUDA/Helpers/ErrorCheck.hpp"

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n) {
		y[i] = x[i] + y[i];
	}
}


int main(void)
{
	int N = 1<<29; //..not sure why 1<<30 gives out of memory error
    float *d_x = NULL, *d_y = NULL;
    CudaSafeCall(hipMalloc((void **)&d_x, sizeof(float)*N));
    CudaSafeCall(hipMalloc((void **)&d_y, sizeof(float)*N));

	// Allocate Unified Memory – accessible from CPU or GPU
    vector<float> vx;
    vector<float> vy;

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		vx.push_back(1.0f);
		vy.push_back(2.0f);
	}

    CudaSafeCall(hipMemcpy(d_x, vx.data(), N*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_y, vy.data(), N*sizeof(float), hipMemcpyHostToDevice));

    //
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, add, 0, N);
    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    cout<<"blockSize: "<<blockSize<<" minGridSize: "<<minGridSize<<" gridSize: "<<gridSize<<endl;

    // calculate theoretical occupancy
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, add, blockSize, 0);

    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    float occupancy = (maxActiveBlocks * blockSize / props.warpSize) /
        (float)(props.maxThreadsPerMultiProcessor /
                props.warpSize);

    printf("Launched blocks of size %d. Theoretical occupancy: %f\n",
            blockSize, occupancy);


	// Run kernel on 1M elements on the GPU
	add<<<gridSize, blockSize>>>(N, d_x, d_y);

	// Wait for GPU to finish before accessing on host
	CudaCheckError(); //.. defined in SPIKE include hipDeviceSynchronize()

    //
    float *h_y = NULL;
    //CudaSafeCall(hipMalloc((void **)&h_y, sizeof(float)*N)); //.. this will core dump. hipMalloc only apply to device array
    //h_y = new float(N*sizeof(float)); //.. core dump for some reason
    h_y = (float*)malloc(N*sizeof(float));
    CudaSafeCall(hipMemcpy(h_y, d_y, N*sizeof(float), hipMemcpyDeviceToHost));
	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(h_y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	CudaSafeCall(hipFree(d_x));
	CudaSafeCall(hipFree(d_y));

	return 0;
}
