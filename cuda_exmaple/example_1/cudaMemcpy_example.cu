// includes, system

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);


int main( int argc, char** argv)
{
    // pointer and dimension for host memory
    int n, dimA;
    float *h_a;
    float *new_a;

    // pointers for device memory
    float *d_a, *d_b;

    // allocate and initialize host memory
    /** Bonus: try using cudaMallocHost in place of malloc **/

    dimA = 8;
    size_t memSize = dimA*sizeof(float);
    //..
    //cudaMallocHost((void**)&h_a, memSize);
    h_a = (float*)malloc(memSize);

    new_a = (float*)malloc(memSize); //.. or use cudaMallocHost(&new_a, memSize); don't use cudaMalloc, it will core-dump
    //cudaMallocHost(&new_a, memSize);  //.. OK
    //cudaMalloc(&new_a, memSize); // core-dump

    for (n=0; n<dimA; n++)
    {
        h_a[n] = (float) n;
    }

    // Part 1 of 5: allocate device memory

    hipMalloc( (void**)&d_a, memSize );
    hipMalloc( (void**)&d_b, memSize );

    // Part 2 of 5: host to device memory copy
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice   );

    // Part 3 of 5: device to device memory copy
    hipMemcpy( d_b, d_a, memSize, hipMemcpyDeviceToDevice );

    // clear host memory
    for (n=0; n<dimA; n++)
    {
        printf("Data in host memory h_a %f\n", h_a[n]);
        //printf("Data in device memory d_a %f\n", d_a[n]);
        //printf("Data in device memory d_b %f\n", d_b[n]);
        h_a[n] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy( new_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (n=0; n<dimA; n++)
    {
        assert(new_a[n] == (float) n);
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree( d_b );
    hipFree( d_a );

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    // Bonus: be sure to use cudaFreeHost for memory allocated with cudaMallocHost

    hipHostFree(h_a);
    //free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("hipHostMalloc is working Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}


























