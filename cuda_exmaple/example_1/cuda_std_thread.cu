// https://stackoverflow.com/questions/57187912/how-to-differentiate-gpu-threads-in-a-single-gpu-for-different-host-cpu-thread
// nvcc cuda_std_thread.cu -o cuda_std_thread -std=c++11
#include <iostream>
#include <math.h>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>


using namespace std;

const unsigned NUM_THREADS = std::thread::hardware_concurrency();  //.. no. of core in the machine

// Kernel function to add the elements of two arrays
__global__
void add_2(int n, float *x, float *y)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < n) {
        y[i] = x[i] + y[i];
    }
}

//
void thread_func(int N, float *x, float *y, int idx_thread)
{
    hipSetDevice(0); //.. set to current GPU. Since I have only one, this is set to 0. Can skip this line since by default, it is set to 0

    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, add_2, 0, N);

    // sample size for each host thread
    int workSize = (N + NUM_THREADS - 1)/NUM_THREADS;

    // Round up according to array size
    gridSize = (workSize + blockSize - 1)/blockSize;

    cout<<"blockSize: "<<blockSize<<" minGridSize: "<<minGridSize<<" gridSize: "<<gridSize<<endl;

    // Run kernel on 1M elements on the GPU
    add_2<<<gridSize, blockSize>>>(workSize, x+idx_thread*workSize, y+idx_thread*workSize);


	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
}

//
int main()
{

	int N = 1<<30;
	float *x, *y;
	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

    //.. begin multithreading ..
    vector<std::thread> t;
    for(int i = 0; i<NUM_THREADS; i++) 
        t.push_back(thread(thread_func, N, x, y, i));

    for(int i = 0; i<NUM_THREADS; i++)
        t[i].join();

	// Check for errors (all values should be 3.0f)
	//float maxError = 0.0f;
	//for (int i = 0; i < N; i++) {
    //    if(!(i%100000000))
    //        std::cout<<i<<" "<<y[i]<<std::endl;
	//	maxError = fmax(maxError, fabs(y[i]-3.0f));
	//}
	//std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

    return 0;
}
