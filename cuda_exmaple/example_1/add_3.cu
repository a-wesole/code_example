#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>
//
#define CUDA_ERROR_CHECK  // enable error check in cuda
#include "/home/wxie/AI/Spike/July_2019/Spike_July_2019/Spike/Backend/CUDA/Helpers/ErrorCheck.hpp"

using namespace std;


// Kernel function to do nested loops
__global__
void add(int max_x, int max_y, float *tot, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
	if(i < max_x && j<max_y) {
        atomicAdd(tot, x[i] + y[j]);
	}
}


int main(void)
{
	int Nx = 1<<15; 
	int Ny = 1<<15; 
    float *d_x = NULL, *d_y = NULL;
    float *d_tot = NULL;
    CudaSafeCall(hipMalloc((void **)&d_x, sizeof(float)*Nx));
    CudaSafeCall(hipMalloc((void **)&d_y, sizeof(float)*Ny));
    CudaSafeCall(hipMalloc((void **)&d_tot, sizeof(float)));

	// Allocate Unified Memory – accessible from CPU or GPU
    vector<float> vx;
    vector<float> vy;

	// initialize x and y arrays on the host
	for (int i = 0; i < Nx; i++) 
		vx.push_back(i);

	for (int i = 0; i < Ny; i++) 
		vy.push_back(i*10);

    //
    float tot = 0;
    for(int i = 0; i<vx.size(); i++)
        for(int j = 0; j<vy.size(); j++) 
            tot += vx[i] + vy[j];

    cout<<"CPU: tot: "<<tot<<endl;


    //
    CudaSafeCall(hipMemcpy(d_x, vx.data(), vx.size()*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_y, vy.data(), vy.size()*sizeof(float), hipMemcpyHostToDevice));

    //
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, add, 0, Nx+Ny);

    //.. bx*by can not go beyond the blockSize, or hardware limit, which is 1024;
    //.. bx*bx = blockSize && bx/by=Nx/Ny, solve the equation
    int bx = sqrt(blockSize*Nx/(float)Ny);
    int by = bx*Ny/(float)Nx;
    dim3 blockSize_3D(bx, by); 
    dim3 gridSize_3D((Nx+bx-1)/bx, (Ny+by+1)/by);   

    cout<<"blockSize: "<<blockSize<<endl;
    cout<<"bx: "<<bx<<" by: "<<by<<" gx: "<<gridSize_3D.x<<" gy: "<<gridSize_3D.y<<endl;

    // calculate theoretical occupancy
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, add, blockSize, 0);

    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    float occupancy = (maxActiveBlocks * blockSize / props.warpSize) /
        (float)(props.maxThreadsPerMultiProcessor /
                props.warpSize);

    printf("Launched blocks of size %d. Theoretical occupancy: %f\n",
            blockSize, occupancy);


	// Run kernel on 1M elements on the GPU
    tot = 0;
	add<<<gridSize_3D, blockSize_3D>>>(Nx, Ny, d_tot, d_x, d_y);

	// Wait for GPU to finish before accessing on host
	CudaCheckError(); //.. defined in SPIKE include hipDeviceSynchronize()

    //
    CudaSafeCall(hipMemcpy(&tot, d_tot, sizeof(float), hipMemcpyDeviceToHost));

    //
    cout<<" GPU: tot: "<<tot<<endl;

	// Free memory
	CudaSafeCall(hipFree(d_x));
	CudaSafeCall(hipFree(d_y));
	CudaSafeCall(hipFree(d_tot));

	return 0;
}
