
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n) {
		y[i] = x[i] + y[i];
	}
}


int main(void)
{
	int N = 1<<30;
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

    //
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, add, 0, N);
    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    cout<<"blockSize: "<<blockSize<<" minGridSize: "<<minGridSize<<" gridSize: "<<gridSize<<endl;

    // calculate theoretical occupancy
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, add, blockSize, 0);

    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    float occupancy = (maxActiveBlocks * blockSize / props.warpSize) /
        (float)(props.maxThreadsPerMultiProcessor /
                props.warpSize);

    printf("Launched blocks of size %d. Theoretical occupancy: %f\n",
            blockSize, occupancy);


	// Run kernel on 1M elements on the GPU
	add<<<gridSize, blockSize>>>(N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}

