#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "add.h"

using namespace std;

//
void ADD::calculate()
{
	int N = 1<<30;
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, add, 0, N);
    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    cout<<"blockSize: "<<blockSize<<" minGridSize: "<<minGridSize<<" gridSize: "<<gridSize<<endl;

    // Run kernel on 1M elements on the GPU
    add<<<gridSize, blockSize>>>(N, x, y);


	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
        //if(!(i%10000))
        //    std::cout<<i<<" "<<y[i]<<std::endl;
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);
}

