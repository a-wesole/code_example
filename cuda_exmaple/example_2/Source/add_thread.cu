#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <thread>
#include <vector>
#include "add_thread.h"

using namespace std;

const unsigned NUM_THREADS = std::thread::hardware_concurrency();  //.. no. of core in the machine

//
void thread_func(int N, float *x, float *y, int idx_thread)
{
    hipSetDevice(0); //.. set to current GPU. Since I have only one, this is set to 0. Can skip this line since by default, it is set to 0

    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, add, 0, N);

    // sample size for each host thread
    int workSize = (N + NUM_THREADS - 1)/NUM_THREADS;

    // Round up according to array size
    gridSize = (workSize + blockSize - 1)/blockSize;

    cout<<"blockSize: "<<blockSize<<" minGridSize: "<<minGridSize<<" gridSize: "<<gridSize<<endl;

    // Run kernel on 1M elements on the GPU
    add<<<gridSize, blockSize>>>(workSize, x+idx_thread*workSize, y+idx_thread*workSize);


    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
}

//
void ADD_thread::calculate()
{

	int N = 1<<30;
	float *x, *y;
	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

    //.. begin multithreading ..
    vector<std::thread> t;
    for(int i = 0; i<NUM_THREADS; i++) 
        t.push_back(thread(thread_func, N, x, y, i));

    for(int i = 0; i<NUM_THREADS; i++)
        t[i].join();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
        //if(!(i%10000))
        //    std::cout<<i<<" "<<y[i]<<std::endl;
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);
}

